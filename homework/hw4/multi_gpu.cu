
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <fstream>
#include <iterator>
#include <vector>
#include <iostream> 
using namespace std;
const int INF = 1000000000;
int n; 
unsigned int m;	
int* Dist;

__global__ void phase_one(int r, int n, int B, int* Dist)
{
    int x = threadIdx.x;
    int y = threadIdx.y;
    int pivot_i = r*B + y;
    int pivot_j = r*B + x;
    extern __shared__ int shared_Dist[];
    // copy to shared memory
    shared_Dist[y*B + x] = (pivot_i<n && pivot_j<n)? Dist[pivot_i*n + pivot_j] : INF;
    __syncthreads();

    // floyd-algo
    #pragma unroll
    for(int k=0; k<B; ++k){
        if(shared_Dist[y*B + x] > shared_Dist[y*B + k] + shared_Dist[k*B + x]){
            shared_Dist[y*B + x] = shared_Dist[y*B + k] + shared_Dist[k*B + x];
        }
        __syncthreads();
    }
    // update global memory
    if(pivot_i<n && pivot_j<n){
        Dist[pivot_i*n + pivot_j] = shared_Dist[y*B + x];
    }
}

__global__ void phase_two(int r, int n, int B, int* Dist)
{
    // pivot
    if(blockIdx.x == r) return;            

    int x = threadIdx.x;
    int y = threadIdx.y;
    int pivot_i = r*B + y;
    int pivot_j = r*B + x;
    extern __shared__ int shared_mem[];
    int* shared_pivot = shared_mem;
    int* shared_Dist = shared_mem + B*B;
    
    // copy pivot to shared memory
    shared_pivot[y*B + x] = (pivot_i<n && pivot_j<n)? Dist[pivot_i*n + pivot_j] : INF;
    __syncthreads();

    int block_i, block_j;
    // same row
    if(blockIdx.y == 0){                    
        block_i = pivot_i;
        block_j = blockIdx.x * B + x; 
    }else{
        // same col                                 
        block_i = blockIdx.x * B + y;
        block_j = pivot_j;
    }

    if(block_i >= n || block_j >= n) return;
    // copy Dist to shared memory
    shared_Dist[y * B + x] = (block_i<n && block_j<n)? Dist[block_i*n + block_j] : INF;
    __syncthreads();

    // same row
    if(blockIdx.y == 0){        
        #pragma unroll
        for(int k=0; k<B; ++k){
            if(shared_Dist[y*B + x] > shared_pivot[y*B + k] + shared_Dist[k*B + x]){
                shared_Dist[y*B + x] = shared_pivot[y*B + k] + shared_Dist[k*B + x];
            }    
            __syncthreads();
        }
    }else{                       
        // same col
        #pragma unroll
        for(int k=0; k<B; ++k){
            if(shared_Dist[y*B + x] > shared_Dist[y*B + k] + shared_pivot[k*B + x]){
                shared_Dist[y*B + x] = shared_Dist[y*B + k] + shared_pivot[k*B + x];
            }     
            __syncthreads();
        }     
    }

    // copy to global memory
    if(block_i<n && block_j<n){
        Dist[block_i*n + block_j] = shared_Dist[y*B + x];
    }
}

__global__ void phase_three(int r, int n, int B, int* Dist, int bias)
{
    int block_Idx_x = blockIdx.x;
    int block_Idx_y = blockIdx.y + bias;
    // pivot or same row, col
    if(block_Idx_x == r || block_Idx_y == r) return;  

    extern __shared__ int shared_mem[];
    int* shared_row = shared_mem;
    int* shared_col = shared_mem + B*B;

    int x = threadIdx.x;
    int y = threadIdx.y;
    int block_i = block_Idx_y*B + y;
    int block_j = block_Idx_x*B + x;
    int row_i = r*B + y;
    int row_j = block_j;
    int col_i = block_i;
    int col_j = r*B + x;
    
    // copy same row,col with pivot to shared memory
    shared_row[y* B + x] = (row_i<n && row_j<n)? Dist[row_i*n + row_j] : INF;
    shared_col[y* B + x] = (col_i<n && col_j<n)? Dist[col_i*n + col_j] : INF;
    __syncthreads();

    if(block_i >= n || block_j >= n) return;

    int target = Dist[block_i * n + block_j];     
    // floyd-algo
    #pragma unroll
    for(int k=0; k<B; ++k){
        if(target > shared_col[y*B + k] + shared_row[k*B + x]){
            target = shared_col[y*B + k] + shared_row[k*B + x];
        }
    }
    // update global memory
    Dist[block_i*n + block_j] = target;
}

void input(char *inFileName)
{
	FILE* file = fopen(inFileName, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    Dist = (int*) malloc((size_t)sizeof(int)*n*n);
    for (int i = 0; i < n; ++ i) {
        for (int j = 0; j < n; ++ j) {
            if (i == j) {
                Dist[i*n + j] = 0;
            } else {
                Dist[i*n + j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++ i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*n + pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char *outFileName)
{
	ofstream ofile;
    ofile.open(outFileName, ios::binary | ios::out);
    for(int i=0; i<n ;i++){
        for(int j=0 ; j<n ; j++){
            ofile.write((char*) &Dist[i*n+j], sizeof(int));
        }
    }
    ofile.close();
}
int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void block_FW(int B)
{
    // 1 or 2
    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    omp_set_num_threads(num_gpus);
    int* device_Dist[num_gpus];
    size_t shared_mem_size = sizeof(int)*n*n;
    
    #pragma omp parallel
    {
        unsigned int num_omp_threads = omp_get_thread_num();
        hipSetDevice(num_omp_threads);
        hipMalloc(&device_Dist[num_omp_threads], shared_mem_size);
        hipMemcpy(device_Dist[num_omp_threads], Dist, shared_mem_size, hipMemcpyHostToDevice);
        
        int num_blocks = ceil(n, B);
        int avg_block = num_blocks/num_gpus;
        int start_round = (num_omp_threads < num_blocks%num_gpus)? (avg_block+1)*num_omp_threads:avg_block*num_omp_threads + num_blocks%num_gpus;
        int block_row = (num_omp_threads < num_blocks%num_gpus)? avg_block+1:avg_block;                  

        dim3 block(B, B);
        dim3 grid_phase1(1, 1);                    
        dim3 grid_phase2(num_blocks, 2);			    	
        dim3 grid_phase3(num_blocks, block_row);	    

        for(int r = 0; r < num_blocks; r++){
            // wait for pivot
            #pragma omp barrier                     
            phase_one<<< grid_phase1, block, sizeof(int) * B*B >>>(r, n, B, device_Dist[num_omp_threads]);
            phase_two<<< grid_phase2, block, sizeof(int) * 2*B*B >>>(r, n, B, device_Dist[num_omp_threads]);
            phase_three<<< grid_phase3, block, sizeof(int) * 2*B*B >>>(r, n, B, device_Dist[num_omp_threads], start_round);
        
            // transfer pivot
            if(r < num_blocks-1){
                // pivot at GPU0: r+1 < start_round + block_row
                // pivot at GPU1: r+1 ≥ start_round
                if(r+1 < start_round+block_row && r+1 >= start_round){
                    // GPU1 r = num_blocks-2 -> n < (r+2)*B
                    int transfer_size = (n >= (r+2)*B)? sizeof(int)*n*B : sizeof(int)*n*(n-(r+1)*B);
                    int transfer_row = r+1;
                    // copy row
                    for(int i = 0; i < num_gpus; i++){   
                        if(i != num_omp_threads)                    
                            hipMemcpy(device_Dist[i] + transfer_row*n*B, device_Dist[num_omp_threads] + transfer_row*n*B, transfer_size, hipMemcpyDeviceToDevice);                    
                    }
                }                                 
            }                        
        }
        
        // GPU0 GPU1 exchange global memory to host
        size_t size = (n >= (start_round+block_row)*B)? sizeof(int)*n*B*block_row : sizeof(int)*n*(n - start_round*B); 
        hipMemcpy(Dist + start_round*n*B, device_Dist[num_omp_threads] + start_round*n*B, size, hipMemcpyDeviceToHost);
    }

    hipFree(device_Dist[0]);
    hipFree(device_Dist[1]);
}

int main(int argc, char* argv[])
{
    input(argv[1]);
    // set block factor for experiment
    int B = argc>3? atoi(argv[3]) : n>32? 32 : n;
    block_FW(B);
    
    output(argv[2]);

    free(Dist);
	return 0;
}