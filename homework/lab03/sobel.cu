
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8
unsigned char *device_s = NULL;
unsigned char *device_t = NULL;
unsigned char *host_s = NULL;       // source image array
unsigned char *host_t = NULL;       // target image array
FILE *fp_s = NULL;                  // source file handler
FILE *fp_t = NULL;                  // target file handler

unsigned int   width, height;       // image width, image height
unsigned int   rgb_raw_data_offset; // RGB raw data offset
unsigned char  bit_per_pixel;       // bit per pixel
unsigned short byte_per_pixel;      // byte per pixel

// bitmap header
unsigned char header[54] = {
	0x42,        // identity : B
	0x4d,        // identity : M
	0, 0, 0, 0,  // file size
	0, 0,        // reserved1
	0, 0,        // reserved2
	54, 0, 0, 0, // RGB data offset
	40, 0, 0, 0, // struct BITMAPINFOHEADER size
	0, 0, 0, 0,  // bmp width
	0, 0, 0, 0,  // bmp height
	1, 0,        // planes
	24, 0,       // bit per pixel
	0, 0, 0, 0,  // compression
	0, 0, 0, 0,  // data size
	0, 0, 0, 0,  // h resolution
	0, 0, 0, 0,  // v resolution 
	0, 0, 0, 0,  // used colors
	0, 0, 0, 0   // important colors
};

// sobel mask (5x5 version)
// Task 2: Put mask[][][] into Shared Memroy
__device__ int dev_mask[MASK_N][MASK_X][MASK_Y] = {
	{{ -1, -4, -6, -4, -1},
	 { -2, -8,-12, -8, -2},
	 {  0,  0,  0,  0,  0},
	 {  2,  8, 12,  8,  2},
	 {  1,  4,  6,  4,  1}},
	{{ -1, -2,  0,  2,  1},
	 { -4, -8,  0,  8,  4},
	 { -6,-12,  0, 12,  6},
	 { -4, -8,  0,  8,  4},
	 { -1, -2,  0,  2,  1}}
};

int read_bmp (const char *fname_s) {
	fp_s = fopen(fname_s, "rb");
	if (fp_s == NULL) {
		printf("fopen fp_s error\n");
		return -1;
	}

	// move offset to 10 to find rgb raw data offset
	fseek(fp_s, 10, SEEK_SET);
	fread(&rgb_raw_data_offset, sizeof(unsigned int), 1, fp_s);

	// move offset to 18 to get width & height;
	fseek(fp_s, 18, SEEK_SET); 
	fread(&width,  sizeof(unsigned int), 1, fp_s);
	fread(&height, sizeof(unsigned int), 1, fp_s);

	// get bit per pixel
	fseek(fp_s, 28, SEEK_SET); 
	fread(&bit_per_pixel, sizeof(unsigned short), 1, fp_s);
	byte_per_pixel = bit_per_pixel / 8;

	// move offset to rgb_raw_data_offset to get RGB raw data
	fseek(fp_s, rgb_raw_data_offset, SEEK_SET);

	// Task 3: Assign host_s to "Pinnned Memory" in CPU malloc doesn't pin
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	// host_s = (unsigned char *) malloc((size_t)width * height * byte_per_pixel);
	// if (host_s == NULL) {
	// 	printf("malloc images_s error\n");
	// 	return -1;
	// }
	hipHostMalloc(&host_s, (size_t)width * height * byte_per_pixel, hipHostMallocDefault);

	// Task 3: Assign host_t to Pinned Memory
	// Hint  : err = cudaMallocHost ( ... )
	//         if (err != CUDA_SUCCESS)
	// host_t = (unsigned char *) malloc((size_t) width * height * byte_per_pixel);
	// if (host_t == NULL) {
	// 	printf("malloc host_t error\n");
	// 	return -1;
	// }
	hipHostMalloc(&host_t, (size_t)width * height * byte_per_pixel, hipHostMallocDefault);

	fread(host_s, sizeof(unsigned char), (size_t)(long) width * height * byte_per_pixel, fp_s);

	return 0;
}

// Executed on the device Callable from the host only
__global__  void sobel (const int width, const int height, const int byte_per_pixel, unsigned char* host_s, unsigned char* host_t) {
	int  x, y, i, v, u;            // for loop counter
	int  R, G, B;                  // color of R, G, B
	double val[MASK_N*3] = {0.0};
	int adjustX, adjustY, xBound, yBound;
	
	// Task 2: Put mask[][][] into Shared Memory
	// Hint  : Please declare it in kernel function
	//         Then use some threads to move data from global memory to shared memory
	//         Remember to __syncthreads() after it's done <WHY?>

	__shared__ int mask[MASK_N][MASK_X][MASK_Y];
	// for(int i = blockIdx.x ; i  <MASK_N; i+=gridDim.x) {
	// 	for(int j = 0 ; j< MASK_X ; j+=)
	// 		for(int k = 0;k<MASK_Y;k++)
	// 			mask[i][j][k] = dev_mask[i][j][k];
	// }
	x = (blockIdx.x * blockDim.x + threadIdx.x)%MASK_N;
	y = (blockIdx.x * blockDim.x + threadIdx.x)/MASK_N%MASK_X;
	i = (blockIdx.x * blockDim.x + threadIdx.x)/MASK_N/MASK_X%MASK_Y;
	mask[x][y][i] = dev_mask[x][y][i];
	__syncthreads();// wait for each thread to copy its elemenet

	if ((blockIdx.x * blockDim.x + threadIdx.x) == height)
	{
		for(x =0;x < MASK_N ; x++)
		{
			for(y=0;y<MASK_X;y++)
			{
				for(i=0;i<MASK_Y;i++)
					printf("%d ",mask[x][y][i]);
				printf("\n");
			}
			
		}	
	}
	// Task 1: Relabel x or y or both into combination of blockIdx, threadIdx ... etc
	// Hint A: We do not have enough threads for each pixels in the image, so what should we do?
	// Hint B: Maybe you can map each y to different threads in different blocks
	y = blockIdx.x * blockDim.x + threadIdx.x;
	// for (y = 0; y < height; ++y) {
	if(y<height){
		for (x = 0; x < width; ++x) {
			for (i = 0; i < MASK_N; ++i) {
				adjustX = (MASK_X % 2) ? 1 : 0;
				adjustY = (MASK_Y % 2) ? 1 : 0;
				xBound = MASK_X /2;
				yBound = MASK_Y /2;

				val[i*3+2] = 0.0;
				val[i*3+1] = 0.0;
				val[i*3] = 0.0;

				for (v = -yBound; v < yBound + adjustY; ++v) {
					for (u = -xBound; u < xBound + adjustX; ++u) {
						if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
							R = host_s[byte_per_pixel * (width * (y+v) + (x+u)) + 2];
							G = host_s[byte_per_pixel * (width * (y+v) + (x+u)) + 1];
							B = host_s[byte_per_pixel * (width * (y+v) + (x+u)) + 0];
							val[i*3+2] += R * mask[i][u + xBound][v + yBound];
							val[i*3+1] += G * mask[i][u + xBound][v + yBound];
							val[i*3+0] += B * mask[i][u + xBound][v + yBound];
						}	
					}
				}
			}

			double totalR = 0.0;
			double totalG = 0.0;
			double totalB = 0.0;
			for (i = 0; i < MASK_N; ++i) {
				totalR += val[i * 3 + 2] * val[i * 3 + 2];
				totalG += val[i * 3 + 1] * val[i * 3 + 1];
				totalB += val[i * 3 + 0] * val[i * 3 + 0];
			}

			totalR = sqrt(totalR) / SCALE;
			totalG = sqrt(totalG) / SCALE;
			totalB = sqrt(totalB) / SCALE;
			const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
			const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
			const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
			host_t[byte_per_pixel * (width * y + x) + 2] = cR;
			host_t[byte_per_pixel * (width * y + x) + 1] = cG;
			host_t[byte_per_pixel * (width * y + x) + 0] = cB;
		}
	}else{
		return;
	}
}

int write_bmp (const char *fname_t) {
	unsigned int file_size;

	fp_t = fopen(fname_t, "wb");
	if (fp_t == NULL) {
		printf("fopen fname_t error\n");
		return -1;
	}

	// file size  
	file_size = width * height * byte_per_pixel + rgb_raw_data_offset;
	header[2] = (unsigned char)(file_size & 0x000000ff);
	header[3] = (file_size >> 8)  & 0x000000ff;
	header[4] = (file_size >> 16) & 0x000000ff;
	header[5] = (file_size >> 24) & 0x000000ff;

	// width
	header[18] = width & 0x000000ff;
	header[19] = (width >> 8)  & 0x000000ff;
	header[20] = (width >> 16) & 0x000000ff;
	header[21] = (width >> 24) & 0x000000ff;

	// height
	header[22] = height &0x000000ff;
	header[23] = (height >> 8)  & 0x000000ff;
	header[24] = (height >> 16) & 0x000000ff;
	header[25] = (height >> 24) & 0x000000ff;

	// bit per pixel
	header[28] = bit_per_pixel;

	// write header
	fwrite(header, sizeof(unsigned char), rgb_raw_data_offset, fp_t);

	// write image
	fwrite(host_t, sizeof(unsigned char), (size_t)(long)width * height * byte_per_pixel, fp_t);

	fclose(fp_s);
	fclose(fp_t);

	return 0;
}

int main(int argc, char **argv) {

    assert(argc == 3);
    const char* input = argv[1];
    const char* output = argv[2];
	read_bmp(input); // 24 bit gray level image
    

	// Task 1: Allocate memory on GPU
	// Hint  : cudaMalloc ()
	//         What do we need to store on GPU? (input image, output image, ...)

	// Task 1: Memory copy from Host to Device (GPU)
	// Hint  : cudaMemcpy ( ... , cudaMemcpyHostToDevice )

	// Task 1: Modify sobel() to CUDA kernel function
	// Hint  : sobel_Kernel <<< ??? , ??? >>> ( ??? );
	// sobel();
	hipMalloc((void**)&device_s, (size_t) width * height * byte_per_pixel);
    hipMalloc((void**)&device_t, (size_t) width * height * byte_per_pixel);
    hipMemcpy(device_s, host_s, (size_t) width * height * byte_per_pixel, hipMemcpyHostToDevice);
	sobel<<<(height/256) + 1, 256>>>(width, height, byte_per_pixel, device_s, device_t);
	// Task 1: Memory Copy from Device (GPU) to Host
	// Hint  : cudaMemcpy ( ... , cudaMemcpyDeviceToHost )
	hipMemcpy(host_t, device_t, (size_t) width * height * byte_per_pixel, hipMemcpyDeviceToHost);
	// Task 1: Free memory on device
	// Hint  : cudaFree ( ... )
	hipFree(device_s);
    hipFree(device_t);
	write_bmp(output);

	// Task 3: Free Pinned memory
	// Hint  : replace free ( ... ) by cudaFreeHost ( ... )
	// free (host_s);
	// free (host_t);
	hipHostFree(host_s);
    hipHostFree(host_t);
}
